/*
 * Vector Addition using CUDA
 *
 * This program demonstrates parallel vector addition on the GPU
 * using CUDA. It creates two random vectors on the CPU, adds them
 * element-wise on the GPU, and stores the result in a third vector.
 * The program includes timing code to measure the execution time of
 * the GPU kernel. The CPU then performs the same vector addition and
 * compares the GPU result with the CPU result to ensure correctness.
 * The program is parameterized by the length of the vectors, specified
 * as a command-line argument.
 *
 * Compilation: $ nvcc vectorAdd.cu -o vectorAdd
 *
 * Execution: $ ./vectorAdd.exe <vector_length>
 *
 * Parameters: <vector_length> - Length of the vectors for vector addition.
 *
 * Profiling with Nvidia Nsight:
 *   1. Compile the code with profiling information:
 *      $ nvcc -lineinfo vectorAdd.cu -o vectorAdd
 *
 *   2. Run the executable with Nvidia Nsight profiling:
 *      $ ncu -o vectorAdd_profile -f ./vectorAdd.exe <vector_length>
 *
 *   3. Analyze the profiling results using Nvidia Nsight Compute:
 *      $ ncu-ui ./vectorAdd_profile.ncu-rep
 *
 * Note: CUDA toolkit must be installed and configured for compilation.
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define DataType double
double start, stop;

/// @brief Starts the timer.
void startTimer()
{
    start = (double)clock();
    start = start / CLOCKS_PER_SEC;
}

/// @brief Stops the timer and print the elapsed time.
void stopTimer(const char* message)
{
    stop = (double)clock();
    stop = stop / CLOCKS_PER_SEC;

    double elapsedTime = (stop - start) * 1.0e3;
    printf("%s: %.6f ms\n", message, elapsedTime);
}

/// @brief Calculates the global index for the current thread and performs 
/// element-wise addition of input vectors, storing the result in _out memory.
__global__ void vecAdd(const DataType* _vecA, const DataType* _vecB, DataType* _out, int _len)
{
    int globalIdx = blockIdx.x * blockDim.x + threadIdx.x;

    if (globalIdx < _len)
    {
        _out[globalIdx] = _vecA[globalIdx] + _vecB[globalIdx];
    }
}

/// @brief Entry point of the program. 
int main(int _argc, char** _argv)
{
    int vectorLength;
    if (_argc != 2)
    {
        fprintf(stderr, "Incorrect input, usage is: %s <vector length>\n", _argv[0]);
        exit(EXIT_FAILURE);
    }

    // Retrieves vector length from the cmd line.
    vectorLength = atoi(_argv[1]);
    const int bytesCount = vectorLength * sizeof(DataType);

    DataType* hostInput1 = (DataType*)malloc(bytesCount);
    DataType* hostInput2 = (DataType*)malloc(bytesCount);
    DataType* hostOutput = (DataType*)malloc(bytesCount);
    DataType* resultRef  = (DataType*)malloc(bytesCount);

    // Fills input vectors with random numbers.
    for (int i = 0; i < vectorLength; ++i)
    {
        hostInput1[i] = rand() / (DataType)RAND_MAX;
        hostInput2[i] = rand() / (DataType)RAND_MAX;
        resultRef[i] = hostInput1[i] + hostInput2[i];
    }

    DataType* deviceInput1;
    DataType* deviceInput2;
    DataType* deviceOutput;

    // Allocatse GPU memory.
    hipMalloc((void**)&deviceInput1, bytesCount);
    hipMalloc((void**)&deviceInput2, bytesCount);
    hipMalloc((void**)&deviceOutput, bytesCount);

    // Profiling scope: Data copy from host to device
    startTimer();
    {
        // Copies memory to the GPU.
        hipMemcpy(deviceInput1, hostInput1, bytesCount, hipMemcpyHostToDevice);
        hipMemcpy(deviceInput2, hostInput2, bytesCount, hipMemcpyHostToDevice);
    }
    stopTimer("Data Copy from Host to Device Time");

    // Computes the 1D thread grid dimensions.
    const int blockSize = 1024;
    const int gridSize = (vectorLength + blockSize - 1) / blockSize;

    // Profiling Scope: CUDA kernel
    startTimer();
    {
        // Runs the GPU Kernel.
        vecAdd<<<gridSize, blockSize>>>(deviceInput1, deviceInput2, deviceOutput, vectorLength);
        hipDeviceSynchronize();
    }
    stopTimer("CUDA Kernel Time");

    // Profiling Scope: Data copy from device to host
    startTimer();
    {
        // Copies the GPU memory back to CPU.
        hipMemcpy(hostOutput, deviceOutput, bytesCount, hipMemcpyDeviceToHost);
    }
    stopTimer("Data Copy from Device to Host Time");

    // Compares result with the reference.
    for (int i = 0; i < vectorLength; ++i)
    {
        if (fabs(hostOutput[i] - resultRef[i]) > 1e-5)
        {
            fprintf(stderr, "Result mismatch found at element %d: %f != %f\n", i, hostOutput[i], resultRef[i]);
            break;
        }
    }

    // Deallocates GPU memory
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    // Deallocates CPU memory
    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
    free(resultRef);

    return EXIT_SUCCESS;
}